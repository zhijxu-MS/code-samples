/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <iostream>
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}
using namespace std;

typedef struct callBackData {
  void *src;
  void *dst;
  void *expected;
  size_t bytes;
} callBackData_t;
void check_accuracy(int *data, int* expected, size_t num)
{
    for(size_t i=0; i<num; i++)
    {
        if(data[i] != expected[i])
        {
            printf("something is wrong, total num is %ld, index is %ld, %d vs %d \n", num, i, data[i], expected[i]);
            return;
        }
    }
    printf("data is equal!\n");
}
void CUDART_CB memcpyHostToHost(void *ptr) {
    printf("in memcpyHostToHost\n");
    auto *info = (callBackData_t*) ptr;
    auto start = chrono::steady_clock::now();
    // checkCuda(cudaMemcpy(info->dst, info->src, info->bytes, cudaMemcpyHostToHost));
    memcpy(info->dst, info->src, info->bytes);
    cout << "H2H time: " << chrono::duration_cast<chrono::microseconds>(chrono::steady_clock::now() - start).count()/1000.0 << "ms" << endl;
    // memcmp(info->dst, info->src, info->bytes);
    // check_accuracy((int*)info->dst, (int*)info->expected, info->bytes/4);
}

int main()
{
  size_t nElements = 1024*1024*1024;
  size_t bytes = nElements * sizeof(float);

  // 2 stream
  int num_streams = 2;
  hipStream_t streams[num_streams];
  for (size_t i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
  }
  // device array
  float *d_a;
  checkCuda( hipMalloc((void**)&d_a, 2*bytes) );
  float *h_expected = (float*)malloc(2*bytes);
  memset(h_expected, 0, 2*bytes);
  for(size_t i=0; i<2*nElements; i++){
    h_expected[i] = i;
  }
  checkCuda(hipMemcpy(d_a, h_expected, 2*bytes, hipMemcpyHostToDevice));
  // host arrays
  float *h_aPageable, *h_bPageable;
  float *h_aPinned, *h_bPinned;
  h_aPageable = (float*)malloc(bytes);
  h_bPageable = (float*)malloc(2*bytes);
  memset(h_aPageable, 0, bytes); // make sure malloc memory is alloc in OS
  memset(h_bPageable, 0, 2*bytes);
  checkCuda( hipHostMalloc((void**)&h_aPinned, bytes, hipHostMallocDefault) );
  checkCuda( hipHostMalloc((void**)&h_bPinned, 2*bytes, hipHostMallocDefault) );
  for (int j=0; j<6; j++){
    //stream 0
    hipMemcpyAsync(h_aPinned, d_a, bytes, hipMemcpyDeviceToHost, streams[0]);
    callBackData_t *host_args1 = new callBackData_t;
    host_args1->src = (void*)h_aPinned;
    host_args1->dst = (void*)h_aPageable;
    host_args1->expected = (void*)h_expected;
    host_args1->bytes = bytes;
    hipLaunchHostFunc(streams[0], memcpyHostToHost, host_args1);
    //stream 1
    hipMemcpyAsync(h_bPageable, d_a, 2*bytes, hipMemcpyDeviceToHost, streams[1]);
    checkCuda(hipDeviceSynchronize());
    // cudaMemcpy(h_bPageable, h_bPinned, 2*bytes, cudaMemcpyHostToHost);
    continue;
    callBackData_t *host_args2 = new callBackData_t;
    host_args2->src = (void*)h_bPinned;
    host_args2->dst = (void*)h_bPageable;
    host_args2->expected = (void*)h_expected;
    host_args2->bytes = bytes;
    hipLaunchHostFunc(streams[1], memcpyHostToHost, host_args2);
  }

    // cleanup
    hipFree(d_a);
    hipHostFree(h_aPinned);
    hipHostFree(h_bPinned);
    free(h_aPageable);
    free(h_bPageable);
    free(h_expected);
    printf("finish main function, everyhing is done\n");
  return 0;
}